#include "hip/hip_runtime.h"
#include "quant_kernel.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define FP16_LIMB_SIZE 16
#define FP16_TYPE uint16_t
#define FP32_TYPE uint32_t //**edited

// defining int32_constants as an array containing 16 elements of type uint32_t
__constant__ uint32_t int32_constants[16];
// same for this case
__constant__ uint64_t int64_constants[2];

// table lookup data, normally we use 5 bit (32 values). But this array size needs to be hardcoded before compilation.
// Use 128 to support up to 7 bits usigned table lookup.(the lookup assume 1 bit automaticcally used for sign => max 8 bits supported by this table)
__constant__ float table_lookup[128];
__constant__ float rounding_hint[128];

#define SIGN_MASK 0x8000                     // 8000 => 1000 0000 0000 0000(16 bits)
#define FLOAT_SIGN_MASK 0x80000000           // => 1000 0000 0000 0000 0000 0000 0000 0000 (32 bits)
#define FLOAT_SIGN_RESET_MASK 0x7FFFFFFF     // => 1 0 bit 31 1 bits
#define SECOND_BIT_MASK 0x4000               // 4000 => 0100 0000 0000 0000
#define FIXED_POSIT_INF 0x0000               // defining the inf for posit
#define FIXED_POSIT_LIMB_ALL_BITS_SET 0xffff // all bits set for a posit
#define SINGLE_PRECISION_BIAS 127            // bias for the single precision number
#define FLOAT_SIZE 32                        // single precision
#define FLOAT_EXPONENT_MASK 0x7f800000       //=> 0111 1111 1000 0000 0000 0000 0000 0000
#define FLOAT_FRACTION_MASK 0x007fffff       //=> 0000 0000 0111 1111 1111 1111 1111 1111
#define FLOAT_SIGN_SHIFT 31
#define FLOAT_EXPONENT_SHIFT 23
#define FLOAT_DENORMAL_EXPONENT -126
#define FLOAT_HIDDEN_BIT_SET_MASK 0x00800000   //=> 0000 0000 1000 0000 0000 0000 0000 0000
#define FLOAT_SIGN_PLUS_EXP_LENGTH_MINUS_ONE 8 // name defined
#define TEMP_TYPE uint64_t                     // defining the temporary type to be unint64_t
#define UNSIGNED_LONG_LONG_SIZE 64             // name defined
#define EDP_ACC_SIZE 63                        // setting the accumulator size
#define FIXED_POSIT_EXP_SHIFT 41               // 64-23
#define FLOAT_EXP_SIGN_SHIFT 30
#define FLOAT_INF 0x7F800000         // 0111 1111 1000 0000 0000 0000 0000 0000
#define FLOAT_SIGN_PLUS_EXP_LENGTH 9 // name defined
#define FIXED_POSIT_LENGTH_PLUS_ONE 17 /

#define GET_MAX(a, b)           \
    ({                          \
        __typeof__(a) _a = (a); \
        __typeof__(b) _b = (b); \
        _a > _b ? _a : _b;      \
    })

#define _G_INFP 32768

union Bits
{
    float f;
    int32_t si;
    uint32_t ui;
};

typedef FP16_TYPE fp16;
typedef FP32_TYPE fp32;
#define _G_FPOSIT_SHIFT_AMOUNT int32_constants[0]
#define _G_MAXREALFP int32_constants[1]
#define _G_MINREALFP int32_constants[2]
#define FPOSIT_EXTRA_BITS_SHIFT int32_constants[3]
#define _G_USEED int32_constants[4]
#define _G_USEED_ZEROS int32_constants[5]
#define FPOSIT_EXPONENT_MASK int32_constants[6]

#define _G_MAXREAL_INT int32_constants[7]
#define _G_MINREAL_INT int32_constants[8]
#define _G_NBITS int32_constants[9]
#define _G_ESIZE int32_constants[10]

#define _G_RSIZE int32_constants[11]
#define FPOSIT_REGIME_MASK int32_constants[12]
#define FPOSIT_FRACTION_MASK int32_constants[13]
#define _G_FSIZE int32_constants[14]
#define _FP_REGIME_BIAS int32_constants[15]
// define regime size

#define FPOSIT_EXTRA_BITS_MASK int64_constants[0]
#define FPOSIT_HALFWAY_BIT_MASK int64_constants[1]

void generate_fixedposit_constants(int nsize, int es, int rf, uint32_t *int32_constants, uint64_t *int64_constants)
{
    // local vars have the same name as global constant vars, confusing but less likely error can happen here.
    // ugly but it's the straightforward conversion from the original #define macroes;
    // todo: make this one less messy

    _G_NBITS = nsize;
    _G_ESIZE = es;
    _G_RSIZE = rf;
    _G_FSIZE = nsize - es - rf - 1;

    if (nsize <= 16 && rf <= 4)
    {
        _G_FPOSIT_SHIFT_AMOUNT = FP16_LIMB_SIZE - nsize;
        _G_MAXREALFP = ((1 << (nsize - 1)) - 1) << _G_FPOSIT_SHIFT_AMOUNT;
        _G_MINREALFP = 0;
        FPOSIT_EXTRA_BITS_SHIFT = UNSIGNED_LONG_LONG_SIZE - nsize + 1;
        _G_USEED = 1 << (1 << es);
        _G_USEED_ZEROS = (1 << es);
        FPOSIT_EXPONENT_MASK = _G_USEED_ZEROS - 1;
        _FP_REGIME_BIAS = (1 << (rf - 1)) - 1;

        _G_MAXREAL_INT = ((((1 << rf) - 1 - _FP_REGIME_BIAS) * (_G_USEED_ZEROS)) + ((1 << es) - 1)) << FLOAT_EXPONENT_SHIFT;
        // _G_MAXREAL_INT = ((_G_USEED_ZEROS * (nsize - 2)) + SINGLE_PRECISION_BIAS) << FLOAT_EXPONENT_SHIFT;
        // this is the maximum float integer that can be represented by the fixed posit.
        int32_constants[8] = (int32_t)(-1 * ((int32_t)int32_constants[15]) * ((int32_t)int32_constants[5])) << 23;

        //_G_MINREAL_INT = (-1 * (_FP_REGIME_BIAS) * (_G_USEED_ZEROS)) << FLOAT_EXPONENT_SHIFT;
        // _G_MINREAL_INT = ((_G_USEED_ZEROS * (2 - nsize)) + SINGLE_PRECISION_BIAS) << FLOAT_EXPONENT_SHIFT;
        // this is the minimum float integer that can be represented by the fixed posit.
        FPOSIT_REGIME_MASK = ((1 << rf) - 1) << (_G_FSIZE + _G_ESIZE);
        FPOSIT_FRACTION_MASK = ((1 << _G_FSIZE) - 1);

        FPOSIT_EXTRA_BITS_MASK = (1UL << (UNSIGNED_LONG_LONG_SIZE - nsize)) - 1;
        FPOSIT_HALFWAY_BIT_MASK = 1UL << (UNSIGNED_LONG_LONG_SIZE - nsize);
    }
    else
    {
        printf("unexpected fposit config\n");
        exit(1);
    }
};

__device__ __inline__ float fixedp16tofp32_gpu(fp16 Fp)
{
    union Bits v;
    // printf("the fraction is: %d \n", Fp);
    bool sign = Fp & SIGN_MASK;
    // printf("the sign is: %d \n", sign);
    Fp = (Fp ^ -sign) + sign; // taking 2s complement if sign is 1, else keeping same
    // printf("the fraction is: %d \n", Fp);
    // int  = _G_NBITS - _G_ESIZE - 1
    v.ui = ((Fp & FPOSIT_REGIME_MASK) >> (_G_ESIZE + _G_FSIZE)) - _FP_REGIME_BIAS; // for regime
    // printf("the current regime is: %d \n", v.ui);
    v.ui = v.ui << _G_ESIZE;
    // printf("the current regime is: %d \n", v.ui);
    // printf("the current float formed is: %f \n", v.f);
    uint32_t exp = (((Fp >> _G_FSIZE) & FPOSIT_EXPONENT_MASK));
    // printf("the current exponent is: %d \n", exp);
    v.ui = (v.ui | exp) + SINGLE_PRECISION_BIAS; // included regime and exponent in the float.
    v.ui = v.ui << 23;
    // printf("the current float formed is: %f \n", v.f);
    uint32_t frac = (Fp & FPOSIT_FRACTION_MASK);
    frac = frac << (23 - _G_FSIZE);
    v.ui = v.ui | frac; // included fraction in the float.
    // printf("the current float formed is: %f \n", v.f);
    v.si ^= (FLOAT_INF ^ v.si) & -(Fp == _G_INFP); // if we had inf in posit, we get inf in float
    v.si ^= (0 ^ v.si) & -(Fp == 0);               // if we had 0 in posit, we get 0 in float

    v.ui |= (sign << FLOAT_SIGN_SHIFT); // putting the sign bit in front
    // printf("the current float formed is: %f \n", v.f);
    return v.f;
}

__device__ __inline__ fp16 fp32tofixedp16_gpu(float f)
{
    fp16 Fp = 0; // initiallising a 16 bit space for the posit
    // printf("FP is : %d \n",Fp);
    union Bits v; // initiallising the union bit space to decode single precision float
    union Bits v2;
    v.f = f; // assigning the bit pattern to the union space
    v2.f = f;
    // printf("the float is: %f \n",f);
    // printf("the value of v is: %f\n", v.f);
    // printf("the value of v is: %d\n", v.ui);
    bool sign = (v.ui & FLOAT_SIGN_MASK); // extracting the sign of the float
    // printf("the sign value is: %d \n", sign);
    v.ui &= 0x7FFFFFFF; // removing the sign from the union space
    // printf("v.ui after and is: %d \n",v.ui);

#ifdef FLOAT_ROUNDING
    uint16_t roundSign = sign << 15;
    if (v.ui > _G_MAXREAL_INT)
        return _G_INFP | roundSign;
    if (v.ui < _G_MINREAL_INT)
        return 0;
#endif
    // corner cases
    Fp ^= (Fp ^ _G_MAXREALFP) & -(v.si >= _G_MAXREAL_INT);              // assign  max real posit value if abs val is >= max real
    Fp ^= (Fp ^ _G_INFP) & -(v.si >= FLOAT_INF);                        // handles infinity
    Fp ^= (Fp ^ _G_MINREALFP) & -(v.si != 0 && v.si <= _G_MINREAL_INT); // assigns  min real posit value if abs val is <= min real
    // printf("FP is : %d \n",Fp);
    // min Fposit exFponent in 16, 3 is -112
    // therefore all the float subnormals will be handled
    // in the Fprevious if statement

    // get exponent sign
    bool exp_sign = !(v.ui >> FLOAT_EXP_SIGN_SHIFT); // getting the sign of the exp, since it has a bias, 1 means + and 0 means -. hence the !
    // printf("the exp_sign value is: %d \n", exp_sign);
    // get regime and exponent
    uint32_t exp = abs((v.si >> FLOAT_EXPONENT_SHIFT) - SINGLE_PRECISION_BIAS); // getting absolute value of the exponent of the float
    // printf("the exponent value is: %d \n", exp);
    TEMP_TYPE regime_and_exp = ((((((exp >> _G_ESIZE) + _FP_REGIME_BIAS))) << (_G_ESIZE)) | (exp & FPOSIT_EXPONENT_MASK)) << _G_FSIZE;
    // printf("the regime and exp value is: %d \n", regime_and_exp);
    // if exponent is negative
    regime_and_exp = ((regime_and_exp ^ -exp_sign) + exp_sign) >> ((exp_sign & !((exp & FPOSIT_EXPONENT_MASK))) & (bool)exp);

    // OBTAINING FRACTION
    TEMP_TYPE frac = v2.ui & FLOAT_FRACTION_MASK;
    frac = frac >> (23 - _G_FSIZE);
    // printf("the fraction is: %d \n", frac);
    // assemble
    fp32 temp_p = frac | regime_and_exp;
    // printf("the assembled value is: %d \n", temp_p);
    // round
    temp_p += (bool)(regime_and_exp & FPOSIT_HALFWAY_BIT_MASK) && ((temp_p & 1) | (regime_and_exp & FPOSIT_EXTRA_BITS_MASK));
    if (_G_NBITS != 16)
        temp_p <<= _G_FPOSIT_SHIFT_AMOUNT;
    // printf("the current temp_p value is: %d \n", temp_p);
    // printf("the current v.si value is: %d \n", v.si);
    // printf("the current minreal_int value is: %d \n", _G_MINREAL_INT);
    // Fp =  (v.f < (-1*_G_MINREAL_INT));
    // printf("the current output value is: %d \n", Fp);
    //Fp ^= (temp_p ^ Fp) & -((v.f < _G_MAXREAL_INT) & (v.f < (-1 * _G_MINREAL_INT)));
    Fp ^= (temp_p ^ Fp) & -((v.f < (float)int32_constants[7]) & (v.f < (-1 * (float)((int32_t)int32_constants[8]))));

    // printf("the current output value is: %d \n", Fp);
    Fp = (Fp ^ -sign) + sign;
    // printf("result from the fp32 to fixed posit 16 function-----> the current output value is: %d \n", Fp);
    return Fp;
}


__global__ void fixed_posit_kernel_nearest(float *input, float *output, float scale, size_t input_size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < input_size)
    {
        float temp_input = input[index] * scale;

        fp16 temp = fp32tofixedp16_gpu(temp_input);
        temp_input = fixedp16tofp32_gpu(temp);

        output[index] = temp_input / scale;
    }
}

// __device__ float new_format_quantize_nearest(float input)
// {
//     float constants[32] = {1.0 / 65536, 1.0 / 32768, 1.0 / 16384, 1.0 / 8192, 1.0 / 4096, 1.0 / 2048, 1.0 / 1024, 1.0 / 512, 1.0 / 256, 1.0 / 128,
//                            3.0 / 256, 1.0 / 64, 5.0 / 256, 3.0 / 128, 7.0 / 256, 1.0 / 32, 9.0 / 256, 5.0 / 128, 3.0 / 64, 7.0 / 128,
//                            1.0 / 16, 9.0 / 128, 5.0 / 64, 3.0 / 32, 7.0 / 64, 1.0 / 8, 9.0 / 64, 3.0 / 16, 1.0 / 4, 3.0 / 8, 1.0 / 2, 1.0};
//     float result = 0.0;
//     if (input != 0.0)
//     {

//         float min_abs_err = 1e5;
//         float min_constant = 0.0;
//         for (int i = 0; i < 32; i++)
//         {
//             float abs_err = fabsf(constants[i] - fabsf(input));
//             if (abs_err < min_abs_err)
//             {
//                 min_abs_err = abs_err;
//                 min_constant = constants[i];
//             }
//         }

//         if (input < 0)
//             result = -min_constant;
//         else
//             result = min_constant;
//     }

//     return result;
// }

// __device__ float act_format_quantize_nearest(float input)
// {

//     float constants[32] = {1.0 / 4096, 1.0 / 2048, 1.0 / 1024, 1.0 / 512, 1.0 / 256, 1.0 / 128, 1.0 / 64, 1.0 / 32, 1.0 / 16, 1.0 / 8, 3.0 / 16,
//                            1.0 / 4, 5.0 / 16, 3.0 / 8, 7.0 / 16, 1.0 / 2, 9.0 / 16, 5.0 / 8, 3.0 / 4, 7.0 / 8, 1.0, 9.0 / 8, 5.0 / 4, 3.0 / 2,
//                            7.0 / 4, 2.0, 9.0 / 4, 3.0, 4.0, 6.0, 8.0, 16.0};
//     float result = 0.0;
//     if (input != 0.0)
//     {

//         float min_abs_err = 1e5;
//         float min_constant = 0.0;
//         for (int i = 0; i < 32; i++)
//         {
//             float abs_err = fabsf(constants[i] - fabsf(input));
//             if (abs_err < min_abs_err)
//             {
//                 min_abs_err = abs_err;
//                 min_constant = constants[i];
//             }
//         }

//         if (input < 0)
//             result = -min_constant;
//         else
//             result = min_constant;
//     }

//     return result;
// }

// __device__ float configurable_table_quantize_nearest(float input, int table_size)
// {

//     float result = 0.0;
//     if (input != 0.0)
//     {

//         float min_abs_err = 1e5;
//         float min_constant = 0.0;
//         for (int i = 0; i < table_size; i++)
//         {
//             float abs_err = fabsf(table_lookup[i] - fabsf(input));
//             if (abs_err < min_abs_err)
//             {
//                 min_abs_err = abs_err;
//                 min_constant = table_lookup[i];
//             }
//         }

//         if (input < 0)
//             result = -min_constant;
//         else
//             result = min_constant;
//     }

//     return result;
// }

// __device__ float configurable_table_quantize_rounding_hint(float input, int table_size)
// {

//     float result = 0.0;
//     if (input != 0.0)
//     {
//         float min_constant = 0.0;
//         for (int i = 0; i < table_size; i++)
//         {
//             // float abs_err = fabs(constants[i] - fabs(input));
//             if (fabsf(input) > rounding_hint[i])
//                 min_constant = table_lookup[i];
//             // printf( " %f %f %f \n", input , rounding_hint[i] , min_constant);
//         }

//         if (input < 0)
//             result = -min_constant;
//         else
//             result = min_constant;
//     }

//     return result;
// }

// // template <typename scalar_t>
// __global__ void newformat_kernel_nearest(float *input, float *output, float scale, size_t input_size)
// {
//     const int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if (index < input_size)
//     {
//         float temp_input = input[index] * scale;

//         temp_input = new_format_quantize_nearest(temp_input);

//         output[index] = temp_input / scale;
//     }
// }

// __global__ void actformat_kernel_nearest(float *input, float *output, float scale, size_t input_size)
// {
//     const int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if (index < input_size)
//     {
//         float temp_input = input[index] * scale;

//         temp_input = act_format_quantize_nearest(temp_input);

//         output[index] = temp_input / scale;
//     }
// }

// __global__ void configurable_table_kernel_nearest(float *input, float *output, float scale, size_t input_size, size_t table_size)
// {
//     const int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if (index < input_size)
//     {
//         float temp_input = input[index] * scale;

//         temp_input = configurable_table_quantize_nearest(temp_input, table_size);

//         output[index] = temp_input / scale;
//     }
// }

// __global__ void configurable_quantize_kernel_rounding_hint(float *input, float *output, float scale, size_t input_size, size_t table_size)
// {
//     const int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if (index < input_size)
//     {
//         float temp_input = input[index] * scale;

//         temp_input = configurable_table_quantize_rounding_hint(temp_input, table_size);

//         output[index] = temp_input / scale;
//     }
// }


void fixed_posit_kernel_nearest_wrapper(float *__restrict__ a,
                                  float *o, int size, int nsize, int es, int rf, float scale, int blockNums, int blockSize)
{

    uint32_t int32_constants_host[16];
    uint64_t int64_constants_host[2];
    generate_fixedposit_constants(nsize, es, rf, int32_constants_host, int64_constants_host);

    hipMemcpyToSymbol(HIP_SYMBOL(int32_constants), &int32_constants_host[0], 16 * sizeof(uint32_t), 0);
    hipMemcpyToSymbol(HIP_SYMBOL(int64_constants), &int64_constants_host[0], 2 * sizeof(uint64_t), 0);

    fixed_posit_kernel_nearest<<<blockNums, blockSize>>>(a,
                                                   o,
                                                   scale,
                                                   size);
}

// void newformat_kernel_nearest_wrapper(float *__restrict__ a,
//                                       float *o, int size, float scale, int blockNums, int blockSize)
// {

//     newformat_kernel_nearest<<<blockNums, blockSize>>>(a,
//                                                        o,
//                                                        scale,
//                                                        size);
// }

// void actformat_kernel_nearest_wrapper(float *__restrict__ a,
//                                       float *o, int size, float scale, int blockNums, int blockSize)
// {

//     actformat_kernel_nearest<<<blockNums, blockSize>>>(a,
//                                                        o,
//                                                        scale,
//                                                        size);
// }

// void configurable_quantize_kernel_nearest_wrapper(float *__restrict__ a,
//                                                   float *o,
//                                                   /*table lookup data*/
//                                                   float *constants,
//                                                   int table_size, int size, float scale, int blockNums, int blockSize)
// {

//     hipMemcpyToSymbol(HIP_SYMBOL(table_lookup), &constants[0], table_size * sizeof(float), 0);
//     configurable_table_kernel_nearest<<<blockNums, blockSize>>>(a,
//                                                                 o,
//                                                                 scale,
//                                                                 size,
//                                                                 table_size);
// }

// void configurable_quantize_kernel_rounding_hint_wrapper(float *__restrict__ a,
//                                                         float *o,
//                                                         /*table lookup data*/
//                                                         float *constants,
//                                                         float *round_hints,
//                                                         int table_size, int size, float scale, int blockNums, int blockSize)
// {

//     hipMemcpyToSymbol(HIP_SYMBOL(table_lookup), &constants[0], table_size * sizeof(float), 0);
//     hipMemcpyToSymbol(HIP_SYMBOL(rounding_hint), &round_hints[0], table_size * sizeof(float), 0);

//     configurable_quantize_kernel_rounding_hint<<<blockNums, blockSize>>>(a,
//                                                                          o,
//                                                                          scale,
//                                                                          size,
//                                                                          table_size);
// }